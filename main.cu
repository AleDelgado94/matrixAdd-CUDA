#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>


const int N = 100;
const int M = 100;

__global__ void matrixAdd(int* A, int* B, int* C){
	//Posicion del thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	int pos = i * N + j;

	if(i < N && j < M && (N*M) <= 1024){
		C[pos] = A[pos] + B[pos];
	}


}

/*
void sumarVectores(int* A, int* B, int* C, int num_elements){
	//Posicion del thread
	//int i = blockIdx.x * blockDim.x + threadIdx.x;


	for(int i=0; i<num_elements; i++){
		C[i] = A[i] + B[i];
	}
}*/

void fError(hipError_t err){
	if(err != hipSuccess){
		printf("Ha ocurrido un error con codigo: %s\n", hipGetErrorString(err));
	}
}


int main(){


	//Reservar espacio en memoria HOST


	int h_A[N][M];
	int h_B[N][M];
	int h_C[N][M];


	//int * h_A = (int*)malloc(num_elements * sizeof(int));
	//int * h_B = (int*)malloc(num_elements * sizeof(int));
	//int * h_C = (int*)malloc(num_elements * sizeof(int));

	/*if(h_A == NULL || h_B == NULL || h_C == NULL){
		printf("Error al reservar memoria para los vectores HOST");
		exit(1);
	}*/





	//Inicializar elementos de los vectores
	for(int i=0; i<N; i++){
		for(int j=0; j<M; j++){
			h_A[i][j] = 1;
			h_B[i][j] = i;
		}
	}

	hipError_t err;

	int size = N * M * sizeof(int);
	//int size_col = M * sizeof(int);

	int * d_A = NULL;


	err = hipMalloc((void **)(&d_A), size);

	fError(err);

	//for(int i=0; i<M; i++){
		//cudaMalloc((void**)(&d_A[i]), size_col);
		hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	//}


	//err = cudaMalloc((void **)*&d_A, size)

	int * d_B = NULL;
	err = hipMalloc((void **)(&d_B), size);
	fError(err);

	//for(int i=0; i<M; i++){
		//cudaMalloc((void**)(&d_B[i]), size_col);
		hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	//}

	int * d_C = NULL;
	err = hipMalloc((void **)(&d_C), size);
	fError(err);

	//for(int i=0; i<M; i++){
		//cudaMalloc((void**)(&d_C[i]), size_col);
		hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
	//}

	//Copiamos a GPU DEVICE
	//err = cudaMemcpy(d_A, h_A, size, cudaMemcpyHostToDevice);
	//err = cudaMemcpy(d_B, h_B, size, cudaMemcpyHostToDevice);
	//err = cudaMemcpy(d_C, h_C, size, cudaMemcpyHostToDevice);

	//int HilosPorBloque = 512;

	int hilos = 256;
	int bloques = (N * M + hilos - 1) / hilos;
	dim3 HilosPorBloque(16,16,1);
	//int BloquesPorGrid = (N * M + HilosPorBloque -1) / HilosPorBloque;

	dim3 BloquesPorGrid(bloques, bloques);

	hipError_t Err;

	//Lanzamos el kernel y medimos tiempos
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);


	matrixAdd<<<BloquesPorGrid, HilosPorBloque>>>(reinterpret_cast<int*>(&d_A), reinterpret_cast<int*>(&d_B), reinterpret_cast<int*>(&d_C));
	Err = hipGetLastError();
	fError(Err);


	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float tiempo_reserva_host;
	hipEventElapsedTime(&tiempo_reserva_host, start, stop);


	printf("Tiempo de suma vectores DEVICE: %f\n", tiempo_reserva_host);

	hipEventDestroy(start);
	hipEventDestroy(stop);


	for(int i=0; i<N*M; i++){
		printf("%d\n", d_C[i]);
		printf("\n");
	}


	//Copiamos a CPU el vector C
	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);



}







